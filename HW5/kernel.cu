#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int maxIterations,
                             int* Md, int width, int height) {
    
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;

    float x = lowerX + Col * stepX;
    float y = lowerY + Row * stepY;
    float z_re = x;
    float z_im = y;

    int i;
    for (i = 0; i < maxIterations; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = x + new_re;
        z_im = y + new_im;
    }

    Md[Row * width + Col] = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* __restrict__ img,
             int resX, int resY, int maxIterations)
{   
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int size = sizeof(int) * resX * resY;
    int* __restrict__ M = (int*)malloc (size);

    int* Md;
    hipMalloc(&Md, size);

    // image: 1600 x 1200
    // limit: (64, 16)
    dim3 dimBlock(16, 16);
    dim3 dimGrid(resX / dimBlock.x, resY / dimBlock.y);

    mandelKernel<<<dimGrid, dimBlock>>>(lowerX, lowerY, stepX, stepY, maxIterations, Md, resX, resY);

    hipMemcpy(M, Md, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < resY; ++i)
    {
        for (int j = 0; j < resX; j += 8)
        {
            img[i * resX + j] = M[i * resX + j];
            img[i * resX + j + 1] = M[i * resX + j + 1];
            img[i * resX + j + 2] = M[i * resX + j + 2];
            img[i * resX + j + 3] = M[i * resX + j + 3];
            img[i * resX + j + 4] = M[i * resX + j + 4];
            img[i * resX + j + 5] = M[i * resX + j + 5];
            img[i * resX + j + 6] = M[i * resX + j + 6];
            img[i * resX + j + 7] = M[i * resX + j + 7];
        }
    }

    free(M);
    hipFree(Md);
}
