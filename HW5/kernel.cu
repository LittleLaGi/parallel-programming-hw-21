#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int maxIterations,
                             int* Md, int width, int height, int pitch) {
    
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;

    float x = lowerX + Col * stepX;
    float y = lowerY + Row * stepY;
    float z_re = x;
    float z_im = y;

    int i;
    for (i = 0; i < maxIterations; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = x + new_re;
        z_im = y + new_im;
    }

    Md[Row * pitch + Col] = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* __restrict__ img,
             int resX, int resY, int maxIterations)
{   
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int* Md;
    int* M;
    size_t pitch;
    hipMallocPitch(&Md, &pitch, resX * sizeof(int), resY);
    hipHostAlloc(&M, resY * pitch, hipHostMallocMapped);

    // image: 1600 x 1200
    // limit: (64, 16)
    dim3 dimBlock(16, 16);
    dim3 dimGrid(resX / dimBlock.x, resY / dimBlock.y);

    mandelKernel<<<dimGrid, dimBlock>>>(lowerX, lowerY, stepX, stepY, maxIterations, Md, resX, resY, pitch / sizeof(int));

    hipMemcpy(M, Md, resY * pitch, hipMemcpyDeviceToHost);

    pitch /= sizeof(int);
    for (int i = 0; i < resY; ++i)
    {
        for (int j = 0; j < resX; j += 8)
        {
            img[i * resX + j] = M[i * pitch + j];
            img[i * resX + j + 1] = M[i * pitch + j + 1];
            img[i * resX + j + 2] = M[i * pitch + j + 2];
            img[i * resX + j + 3] = M[i * pitch + j + 3];
            img[i * resX + j + 4] = M[i * pitch + j + 4];
            img[i * resX + j + 5] = M[i * pitch + j + 5];
            img[i * resX + j + 6] = M[i * pitch + j + 6];
            img[i * resX + j + 7] = M[i * pitch + j + 7];
        }
    }
    
    hipHostFree(M);
    hipFree(Md);
}
